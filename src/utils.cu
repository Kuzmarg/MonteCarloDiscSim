#include "hip/hip_runtime.h"
#include "utils.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__host__ __device__ void get_square_vertices(double x, double y, double s, double cos_t, double sin_t, double *vertices) {
    double x_rot, y_rot;
    for (int i = 0; i < 4; i++) {
        rotate_point((i % 2) ? -s/2 : +s/2, (i < 2) ? s/2 : -s/2, cos_t, sin_t, &x_rot, &y_rot);
        vertices[2*i] = x + x_rot;
        vertices[2*i + 1] = y + y_rot;
    } 
}

__host__ __device__ void quaternion_to_angle(double qz, double qw, double *sin_t, double *cos_t) {
    *cos_t = 2*qw*qw - 1;
    *sin_t = 2*qw*qz;
}

__host__ __device__ void rotate_point(double x, double y, double cos_t, double sin_t, double *x_rot, double *y_rot) {
    *x_rot = x*cos_t - y*sin_t;
    *y_rot = x*sin_t + y*cos_t;
}

////////////////////////////////////////////////////////////////////////////////

__host__ double rand_double(double high) {
    return ((double)rand() * high) / (double)RAND_MAX;
}

__host__ int rand_int(int high) {
    return rand() % high;
}

////////////////////////////////////////////////////////////////////////////////

__global__ void rand_init_kernel(hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, idx, 0, &state[idx]);
}

__device__ double rand_double_cuda(double high, hiprandState *state) {
    return hiprand_uniform(state) * high;
}

__device__ int rand_int_cuda(int high, hiprandState *state) {
    return (int)(hiprand_uniform(state) * high);
}

////////////////////////////////////////////////////////////////////////////////

__host__ __device__ double distance(const Particle* p1, const Particle* p2, const Config* config) {
    double dx = fmin(fabs(p1->x - p2->x), fabs(config->Lx - fabs(p1->x - p2->x)));
    double dy = fmin(fabs(p1->y - p2->y), fabs(config->Ly - fabs(p1->y - p2->y)));
    return sqrt(pow(dx, 2) + pow(dy, 2));
}

__host__ __device__ double distance_patch(const Patch *p1, const Patch *p2, const Config *config) {
    double dx = fmin(fabs(p1->x - p2->x), fabs(config->Lx - fabs(p1->x - p2->x)));
    double dy = fmin(fabs(p1->y - p2->y), fabs(config->Ly - fabs(p1->y - p2->y)));
    return sqrt(pow(dx, 2) + pow(dy, 2));
}

////////////////////////////////////////////////////////////////////////////////

__host__ int write_xyz(const char* filename, const Config* config, const CellLinkedGrid* cll) {
    FILE *file = fopen(filename, "w");
    if (file == NULL) return 1;
    // 92 is the size of a line for each particle, 100 is for the header
    char *file_string = (char *)malloc((92*config->N*(config->num_patches + 1) + 100) * sizeof(char));
    size_t offset = 0;

    char type = config->type == CIRCLE ? 'C' : 'S';
    offset += sprintf(file_string + offset, "%d\n", config->N + config->N * config->num_patches);
    offset += sprintf(file_string + offset, "Properties=species:S:1:pos:R:3:orientation:R:4:aspherical_shape:R:3\n");
    for (size_t idx = 0; idx < config->N; idx++) {
        offset += sprintf(file_string + offset, "%c %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f\n",
            type,
            cll->particles[idx].x,
            cll->particles[idx].y,
            0.0,
            0.0,
            0.0,
            cll->particles[idx].qz,
            cll->particles[idx].qw,
            config->size/2,
            config->size/2,
            0.2
        );

        // Write patches
        for (size_t j = 0; j < config->num_patches; j++) {
            double cos_t = 2*cll->particles[idx].qw*cll->particles[idx].qw - 1;
            double sin_t = 2*cll->particles[idx].qw*cll->particles[idx].qz;
            double x_rot, y_rot;
            rotate_point(config->patch_coordinates[j].x, config->patch_coordinates[j].y, cos_t, sin_t, &x_rot, &y_rot);
            offset += sprintf(file_string + offset, "P %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f %8.3f\n",
                cll->particles[idx].x + x_rot,
                cll->particles[idx].y + y_rot,
                0.0,
                0.0,
                0.0,
                cll->particles[idx].qz,
                cll->particles[idx].qw,
                config->patch_size/2,
                config->patch_size/2,
                0.2
            );
        }
    }
    fprintf(file, "%s", file_string);
    fclose(file);
    return 0;
}
