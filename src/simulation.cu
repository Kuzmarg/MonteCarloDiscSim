#include "hip/hip_runtime.h"
#include "simulation.h"
#include "cell.h"
#include "utils.h"
#include "movement.h"

#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

int simulate_random(Config *config) {
    CellLinkedGrid cll;
    random_gen(config, &cll);
    char filename[256];
    sprintf(filename, "%s/000000.xyz", config->output_folder);
    int write_code = write_xyz(filename, config, &cll);
    if (write_code) return 1;
    for (int i = 1; i <= config->num_steps; i++) {
        for (size_t idx = 0; idx < config->N; idx++) {
            size_t move_idx = rand_int(config->N);
            random_move(&cll.particles[move_idx], config, &cll);
        }

        if (i % config->save_interval == 0) {
            sprintf(filename, "%s/%06d.xyz", config->output_folder, i);
            write_code = write_xyz(filename, config, &cll);
            if (write_code) return 1;
            printf("Iteration %d finished\n", i);
        }
    }
    cll_free(&cll);
    return 0;
}

int simulate_random_cuda(Config *config) {
    CellLinkedGrid cll, cll_cuda;
    random_gen(config, &cll);
    cll_cuda = cll;
    cll_copy_cuda(&cll, &cll_cuda, config);
    
    Config *d_config;
    CellLinkedGrid *d_cll;
    hipMalloc((void**)&d_config, sizeof(Config));
    hipMemcpy(d_config, config, sizeof(Config), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cll, sizeof(CellLinkedGrid));
    hipMemcpy(d_cll, &cll_cuda, sizeof(CellLinkedGrid), hipMemcpyHostToDevice);

    // Generate initial configuration
    config->Nx_cuda = min(config->Nx_cuda, cll.n_x / 6);
    config->Ny_cuda = min(config->Ny_cuda, cll.n_y / 6);

    // Save initial configuration
    char filename[256];
    sprintf(filename, "%s/000000.xyz", config->output_folder);
    int write_code = write_xyz(filename, config, &cll);
    if (write_code) return 1;
    int n_moves = (int)ceil((double)(config->N) / (config->Nx_cuda * config->Ny_cuda * 4));

    // Initialize random number generator
    hiprandState *d_states;
    hipMalloc((void**)&d_states, config->Nx_cuda * config->Ny_cuda * sizeof(hiprandState));
    rand_init_kernel<<<config->Ny_cuda, config->Nx_cuda>>>(d_states);
    hipDeviceSynchronize();

    // Allocate memory for overlaps and energies
    int *shared_states;
    float *shared_energies;
    Particle *shared_particles;
    hipMalloc((void**)&shared_states, config->Nx_cuda * config->Ny_cuda * 9 * sizeof(int));
    hipMalloc((void**)&shared_particles, config->Nx_cuda * config->Ny_cuda * 2 * sizeof(Particle));
    hipMalloc((void**)&shared_energies, config->Nx_cuda * config->Ny_cuda * 9 * sizeof(float));

    // Simulation steps
    for (int i = 1; i <= config->num_steps; i++) {
        for (int j = 0; j < n_moves; j++) {
            for (int stage = 0; stage < 4; stage++) {
                random_move_kernel<<<config->Ny_cuda * config->Nx_cuda, 9>>>(d_config, d_cll, d_states,
                    shared_states, shared_particles, shared_energies, stage);
                hipDeviceSynchronize();
            }
        }

        if (i % config->save_interval == 0) {
            hipMemcpy(&cll_cuda, d_cll, sizeof(CellLinkedGrid), hipMemcpyDeviceToHost);
            cll_copy_host(&cll, &cll_cuda, config);
            sprintf(filename, "%s/%06d.xyz", config->output_folder, i);
            write_code = write_xyz(filename, config, &cll);
            if (write_code) return 1;
            printf("Iteration %d finished\n", i);
        }
    }
    hipFree(shared_states);
    hipFree(shared_particles);
    hipFree(shared_energies);

    cll_free(&cll);
    cll_free_cuda(&cll_cuda);
    hipFree(d_config);
    hipFree(d_cll);
    hipFree(d_states);
    return 0;
}
