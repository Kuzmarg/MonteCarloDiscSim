#include "hip/hip_runtime.h"
#include "movement.h"
#include "cell.h"
#include "utils.h"

#include <stdio.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

__host__ void random_move(Particle *p, const Config *config, CellLinkedGrid *cll) {
    double rand_sample = rand_double(1);
    Particle moved_particle = *p;
    if (rand_sample < 0.5) {
        double angle = rand_double(2 * M_PI);
        double distance = rand_double(config->max_translation);
        moved_particle.x += distance * cos(angle);
        moved_particle.y += distance * sin(angle);
        moved_particle.x = fmod(moved_particle.x + config->Lx, config->Lx);
        moved_particle.y = fmod(moved_particle.y + config->Ly, config->Ly);
    } else {
        double angle = rand_double(config->max_rotation);
        moved_particle.qw = cos(angle/2);
        moved_particle.qz = sin(angle/2);
    }

    cll_remove_point(p, cll);
    if (!cll_check_overlap(&moved_particle, cll, config)) {
        double delta_energy = cll_patch_energy(&moved_particle, cll, config) - cll_patch_energy(p, cll, config);
        rand_sample = rand_double(1);
        if (delta_energy > 0 && rand_sample > exp(-delta_energy)) {
            cll_add_point(p, cll);
            return;
        }
        *p = moved_particle;
    }
    cll_add_point(p, cll);
    return;
}

__global__ void random_move_kernel(const Config *config, CellLinkedGrid *cll, hiprandState* states, int stage) {
    unsigned rand_idx = threadIdx.x;
    unsigned int cells_per_thread_x = cll->n_x / config->Nx_cuda, remainder_x = cll->n_x % config->Nx_cuda;
    unsigned int cells_per_thread_y = cll->n_y / config->Ny_cuda, remainder_y = cll->n_y % config->Ny_cuda;

    unsigned int x_idx = threadIdx.x / config->Ny_cuda;
    unsigned int y_idx = threadIdx.x % config->Ny_cuda;
    unsigned int x_start = x_idx * cells_per_thread_x + min(x_idx, remainder_x);
    unsigned int x_end = (x_idx + 1) * cells_per_thread_x + min(x_idx + 1, remainder_x);
    unsigned int y_start = y_idx * cells_per_thread_y + min(y_idx, remainder_y);
    unsigned int y_end = (y_idx + 1) * cells_per_thread_y + min(y_idx + 1, remainder_y);

    unsigned int X0, X1, Y0, Y1; // start and end cell coordinates for this step
    switch (stage) {
    case 0:
        X0 = x_start;
        X1 = (x_start + x_end) / 2;
        Y0 = y_start;
        Y1 = (y_start + y_end) / 2;
        break;
    case 1:
        X0 = (x_start + x_end) / 2;
        X1 = x_end;
        Y0 = y_start;
        Y1 = (y_start + y_end) / 2;
        break;
    case 2:
        X0 = x_start;
        X1 = (x_start + x_end) / 2;
        Y0 = (y_start + y_end) / 2;
        Y1 = y_end;
        break;
    case 3:
        X0 = (x_start + x_end) / 2;
        X1 = x_end;
        Y0 = (y_start + y_end) / 2;
        Y1 = y_end;
        break;
    }

    unsigned int num_particles = 0;
    for (unsigned int x = X0; x < X1; x++)
        for (unsigned int y = Y0; y < Y1; y++)
            num_particles += cll->head[x * cll->n_y + y];
    if (num_particles == 0) return;
    unsigned int rand_particle = rand_int_cuda(num_particles, &states[rand_idx]);
    unsigned int cell_idx = 0;
    unsigned int particle_idx = 0;
    for (unsigned int x = X0; x < X1; x++) {
        for (unsigned int y = Y0; y < Y1; y++) {
            unsigned int cell_count = cll->head[x * cll->n_y + y];
            if (rand_particle < cell_count) {
                cell_idx = x * cll->n_y + y;
                particle_idx = rand_particle;
                break;
            }
            rand_particle -= cell_count;
        }
        if (rand_particle < cll->head[cell_idx]) break;
    }
    Particle p = cll->cells[cell_idx * cll->max_particles + particle_idx];
    Particle moved_particle = p;
    double rand_sample = rand_double_cuda(1, &states[rand_idx]);
    if (rand_sample < 0.5) {
        double angle = rand_double_cuda(2 * M_PI, &states[rand_idx]);
        double distance = rand_double_cuda(config->max_translation, &states[rand_idx]);
        moved_particle.x += distance * cos(angle);
        moved_particle.y += distance * sin(angle);
        moved_particle.x = fmod(moved_particle.x + config->Lx, config->Lx);
        moved_particle.y = fmod(moved_particle.y + config->Ly, config->Ly);
    } else {
        double angle = rand_double_cuda(config->max_rotation, &states[rand_idx]);
        moved_particle.qw = cos(angle/2);
        moved_particle.qz = sin(angle/2);
    }
    cll_remove_point(&p, cll);
    if (!cll_check_overlap(&moved_particle, cll, config)) {
        double delta_energy = cll_patch_energy(&moved_particle, cll, config) - cll_patch_energy(&p, cll, config);
        rand_sample = rand_double_cuda(1, &states[rand_idx]);
        if (delta_energy > 0 && rand_sample > exp(-delta_energy)) {
            cll_add_point(&p, cll);
            return;
        }
        p = moved_particle;
    }
    cll_add_point(&p, cll);
}
