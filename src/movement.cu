#include "hip/hip_runtime.h"
#include "movement.h"
#include "cell.h"
#include "utils.h"

#include <stdio.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

__host__ void random_move(Particle *p, const Config *config, CellLinkedGrid *cll) {
    double rand_sample = rand_double(1);
    Particle moved_particle = *p;
    if (rand_sample < 0.5) {
        double angle = rand_double(2 * M_PI);
        double distance = rand_double(config->max_translation);
        moved_particle.x += distance * cos(angle);
        moved_particle.y += distance * sin(angle);
        moved_particle.x = fmod(moved_particle.x + config->Lx, config->Lx);
        moved_particle.y = fmod(moved_particle.y + config->Ly, config->Ly);
    } else {
        double angle = rand_double(config->max_rotation);
        moved_particle.qw = cos(angle/2);
        moved_particle.qz = sin(angle/2);
    }

    cll_remove_point(p, cll);
    if (!cll_check_overlap(&moved_particle, cll, config)) {
        double delta_energy = cll_patch_energy(&moved_particle, cll, config) - cll_patch_energy(p, cll, config);
        rand_sample = rand_double(1);
        if (delta_energy > 0 && rand_sample > exp(-delta_energy)) {
            cll_add_point(p, cll);
            return;
        }
        *p = moved_particle;
    }
    cll_add_point(p, cll);
    return;
}

__global__ void random_move_kernel(const Config *config, CellLinkedGrid *cll, hiprandState* states, int stage) {
    unsigned int thread_id = threadIdx.x;
    unsigned int cells_per_thread_x = cll->n_x / config->Nx_cuda, remainder_x = cll->n_x % config->Nx_cuda;
    unsigned int cells_per_thread_y = cll->n_y / config->Ny_cuda, remainder_y = cll->n_y % config->Ny_cuda;

    unsigned int x_idx = threadIdx.x / config->Ny_cuda;
    unsigned int y_idx = threadIdx.x % config->Ny_cuda;
    unsigned int x_start = x_idx * cells_per_thread_x + min(x_idx, remainder_x);
    unsigned int x_end = (x_idx + 1) * cells_per_thread_x + min(x_idx + 1, remainder_x);
    unsigned int y_start = y_idx * cells_per_thread_y + min(y_idx, remainder_y);
    unsigned int y_end = (y_idx + 1) * cells_per_thread_y + min(y_idx + 1, remainder_y);

    unsigned int X0, X1, Y0, Y1; // start and end cell coordinates for this step
    X0 = x_start + (x_end - x_start) / 2 * (stage % 2);
    Y0 = y_start + (y_end - y_start) / 2 * (stage / 2);
    X1 = (x_start + x_end) / 2 + (x_end - x_start) / 2 * (stage % 2);
    Y1 = (y_start + y_end) / 2 + (y_end - y_start) / 2 * (stage / 2);

    unsigned int rand_cell = rand_int_cuda((X1 - X0) * (Y1 - Y0), &states[thread_id]);
    unsigned int cell_x = X0 + rand_cell / (Y1 - Y0);
    unsigned int cell_y = Y0 + rand_cell % (Y1 - Y0);
    unsigned int cell_idx = cell_x * cll->n_y + cell_y;
    unsigned int cell_size = cll->head[cell_idx];
    if (cell_size == 0) return;
    unsigned int rand_particle = rand_int_cuda(cell_size, &states[thread_id]);
    Particle p = cll->cells[cell_idx * cll->max_particles + rand_particle];
    Particle moved_particle = p;

    double rand_sample = rand_double_cuda(1, &states[thread_id]);
    if (rand_sample < 0.5) {
        double angle = rand_double_cuda(2 * M_PI, &states[thread_id]);
        double distance = rand_double_cuda(config->max_translation, &states[thread_id]);
        moved_particle.x += distance * cos(angle);
        moved_particle.y += distance * sin(angle);
        moved_particle.x = fmod(moved_particle.x + config->Lx, config->Lx);
        moved_particle.y = fmod(moved_particle.y + config->Ly, config->Ly);
    } else {
        double angle = rand_double_cuda(config->max_rotation, &states[thread_id]);
        moved_particle.qw = cos(angle/2);
        moved_particle.qz = sin(angle/2);
    }
    cll_remove_point(&p, cll);
    if (cll_check_overlap(&moved_particle, cll, config)) {
        cll_add_point(&p, cll);
        return;
    }
    
    double delta_energy = cll_patch_energy(&moved_particle, cll, config) - cll_patch_energy(&p, cll, config);
    rand_sample = rand_double_cuda(1, &states[thread_id]);
    if (delta_energy > 0 && rand_sample > exp(-delta_energy)) {
        cll_add_point(&p, cll);
        return;
    }
    p = moved_particle;
    cll_add_point(&p, cll);
}
